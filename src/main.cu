#include "EventManager.h"
#include "fluid_solver.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define SIZE 84

#define IX(i, j, k) ((i) + (M + 2) * ((j) + (N + 2) * (k)))

#define ALIGNED_ARRAY_FLOAT(s,a) static_cast<float*>(std::aligned_alloc(a, s * sizeof(float)))

// Globals for the grid size
static int M = SIZE;
static int N = SIZE;
static int O = SIZE;
static float dt = 0.1f;      // Time delta
static float diff = 0.0001f; // Diffusion constant
static float visc = 0.0001f; // Viscosity constant

// Fluid simulation arrays
static float *u, *v, *w, *u_prev, *v_prev, *w_prev;
static float *dens, *dens_prev;

//Fluid GPU simulation arryas
static float *d_u, *d_v, *d_w, *d_u_prev, *d_v_prev, *d_w_prev;
static float *d_dens, *d_dens_prev;

// Function to allocate simulation data
int allocate_data() {
  int size = (M + 2) * (N + 2) * (O + 2);
  u = new float[size];
  v = new float[size];
  w = new float[size];
  u_prev = new float[size];
  v_prev = new float[size];
  w_prev = new float[size];
  dens = new float[size];
  dens_prev = new float[size];
  if (!u || !v || !w || !u_prev || !v_prev || !w_prev || !dens || !dens_prev) {
    std::cerr << "Cannot allocate memory" << std::endl;
    return 0;
  }

  // Allocate memory on the GPU
  hipMalloc(&d_u, size * sizeof(float));
  hipMalloc(&d_v, size * sizeof(float));
  hipMalloc(&d_w, size * sizeof(float));
  hipMalloc(&d_u_prev, size * sizeof(float));
  hipMalloc(&d_v_prev, size * sizeof(float));
  hipMalloc(&d_w_prev, size * sizeof(float));
  hipMalloc(&d_dens, size * sizeof(float));
  hipMalloc(&d_dens_prev, size * sizeof(float));

  if(!d_u || !d_v || !d_w || !d_u_prev || !d_v_prev || !d_w_prev || !d_dens || !d_dens_prev) {
    std::cerr << "Cannot allocate memory on the GPU" << std::endl;
    return 0;
  }

  return 1;
}
// Function to clear the data (set all to zero)
void clear_data() {
  int size = (M + 2) * (N + 2) * (O + 2);
  for (int i = 0; i < size; i++) {
    u[i] = v[i] = w[i] = u_prev[i] = v_prev[i] = w_prev[i] = dens[i] =
        dens_prev[i] = 0.0f;
  }

  // Copy data to the GPU
  hipMemcpy(d_u, 0, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v, 0, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_w, 0, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_u_prev, 0, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v_prev, 0, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_w_prev, 0, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dens, 0, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dens_prev, 0, size * sizeof(float), hipMemcpyHostToDevice);

}
// Free allocated memory
void free_data() {
  delete[] u;
  delete[] v;
  delete[] w;
  delete[] u_prev;
  delete[] v_prev;
  delete[] w_prev;
  delete[] dens;
  delete[] dens_prev;

  hipFree(d_u);
  hipFree(d_v);
  hipFree(d_w);
  hipFree(d_u_prev);
  hipFree(d_v_prev);
  hipFree(d_w_prev);
  hipFree(d_dens);
  hipFree(d_dens_prev);
}

// Apply events (source or force) for the current timestep
void apply_events(const std::vector<Event>& events) {
    int i = M / 2, j = N / 2, k = O / 2;
    int index = IX(i, j, k); 

    for (const auto& event : events) {
        if (event.type == ADD_SOURCE) {
            dens[index] = event.density;
        } else if (event.type == APPLY_FORCE) {
            u[index] = event.force.x;
            v[index] = event.force.y;
            w[index] = event.force.z;
        }
        std::cout << "Event at timestep " << event.type << " applied " << std::endl;
    }
}


// Function to sum the total density
float sum_density() {
  float total_density = 0.0f;
  int size = (M + 2) * (N + 2) * (O + 2);
  for (int i = 0; i < size; i++) {
    total_density += dens[i];
  }
  return total_density;
}

// Simulation loop
void simulate(EventManager &eventManager, int timesteps) {
  for (int t = 0; t < timesteps; t++) {
    // Get the events for the current timestep
    std::vector<Event> events = eventManager.get_events_at_timestamp(t);

    // Apply events to the simulation
    apply_events(events);

    //Copy data to GPU
    hipMemcpy(d_u, u, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_w, w, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_u_prev, u_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_v_prev, v_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_w_prev, w_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dens, dens, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dens_prev, dens_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);

    // Perform the simulation steps
    vel_step(M, N, O, d_u, d_v, d_w, d_u_prev, d_v_prev, d_w_prev, visc, dt);
    dens_step(M, N, O, d_dens, d_dens_prev, d_u, d_v, d_w, diff, dt);

    //Copy data back to CPU
    hipMemcpy(u, d_u, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(v, d_v, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(w, d_w, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(u_prev, d_u_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(v_prev, d_v_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(w_prev, d_w_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(dens, d_dens, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(dens_prev, d_dens_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyDeviceToHost);
  }
}

int main() {
  // Initialize EventManager
  EventManager eventManager;
  eventManager.read_events("events.txt");

  // Get the total number of timesteps from the event file
  int timesteps = eventManager.get_total_timesteps();

  // Allocate and clear data
  if (!allocate_data())
    return -1;
  clear_data();

  // Run simulation with events
  simulate(eventManager, timesteps);

  // Print total density at the end of simulation
  float total_density = sum_density();
  std::cout << "Total density after " << timesteps
            << " timesteps: " << total_density << std::endl;

  // Free memory
  free_data();

  return 0;
}