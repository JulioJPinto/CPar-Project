#include "hip/hip_runtime.h"
#include "EventManager.h"
#include "fluid_solver.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define SIZE 84

#define IX(i, j, k) ((i) + (M + 2) * ((j) + (N + 2) * (k)))

#define ALIGNED_ARRAY_FLOAT(s,a) static_cast<float*>(std::aligned_alloc(a, s * sizeof(float)))

// Globals for the grid size
static int M = SIZE;
static int N = SIZE;
static int O = SIZE;
static float dt = 0.1f;      // Time delta
static float diff = 0.0001f; // Diffusion constant
static float visc = 0.0001f; // Viscosity constant

// Fluid simulation arrays
float *u, *v, *w, *u_prev, *v_prev, *w_prev;
float *dens, *dens_prev;

//Fluid GPU simulation arryas
float *d_u, *d_v, *d_w, *d_u_prev, *d_v_prev, *d_w_prev;
float *d_dens, *d_dens_prev;

// Function to allocate simulation data
int allocate_data() {
  int size = (M + 2) * (N + 2) * (O + 2);
  u = new float[size];
  v = new float[size];
  w = new float[size];
  u_prev = new float[size];
  v_prev = new float[size];
  w_prev = new float[size];
  dens = new float[size];
  dens_prev = new float[size];
  if (!u || !v || !w || !u_prev || !v_prev || !w_prev || !dens || !dens_prev) {
    std::cerr << "Cannot allocate memory" << std::endl;
    return 0;
  }

  // Allocate memory on the GPU
  hipMalloc(&d_u, size * sizeof(float));
  hipMalloc(&d_v, size * sizeof(float));
  hipMalloc(&d_w, size * sizeof(float));
  hipMalloc(&d_u_prev, size * sizeof(float));
  hipMalloc(&d_v_prev, size * sizeof(float));
  hipMalloc(&d_w_prev, size * sizeof(float));
  hipMalloc(&d_dens, size * sizeof(float));
  hipMalloc(&d_dens_prev, size * sizeof(float));

  if(!d_u || !d_v || !d_w || !d_u_prev || !d_v_prev || !d_w_prev || !d_dens || !d_dens_prev) {
    std::cerr << "Cannot allocate memory on the GPU" << std::endl;
    return 0;
  }

  return 1;
}
// Function to clear the data (set all to zero)
void clear_data() {
  int size = (M + 2) * (N + 2) * (O + 2);
  for (int i = 0; i < size; i++) {
    u[i] = v[i] = w[i] = u_prev[i] = v_prev[i] = w_prev[i] = dens[i] =
        dens_prev[i] = 0.0f;
  }
}
// Free allocated memory
void free_data() {
  delete[] u;
  delete[] v;
  delete[] w;
  delete[] u_prev;
  delete[] v_prev;
  delete[] w_prev;
  delete[] dens;
  delete[] dens_prev;

  hipFree(d_u);
  hipFree(d_v);
  hipFree(d_w);
  hipFree(d_u_prev);
  hipFree(d_v_prev);
  hipFree(d_w_prev);
  hipFree(d_dens);
  hipFree(d_dens_prev);
}

// Kernel to apply density source
__global__ void apply_density_kernel(float *d, int index, float density) {
  d[index] = density;
}

// Kernel to apply forces
__global__ void apply_forces_kernel(float *x, float *y, float *z, int index, float fx, float fy, float fz) {
  x[index] = fx;
  y[index] = fy;
  z[index] = fz;
}

// Apply events (source or force) for the current timestep
void apply_events(const std::vector<Event> &events) {
  bool dens = false; 
  bool force = false;
  int index = IX(M / 2, N / 2, O / 2);

  float density = 0.0f, fx = 0.0f, fy = 0.0f, fz = 0.0f;

  for (const auto &event : events) {
    if (event.type == ADD_SOURCE) {
        dens = true;
        density = event.density;
    } else if (event.type == APPLY_FORCE) {
        force = true;
        fx = event.force.x;
        fy = event.force.y;
        fz = event.force.z;
    }
  }

  if (dens) {
      apply_density_kernel<<<1, 1>>>(d_dens, index, density);
  }

  if (force) {
      apply_forces_kernel<<<1, 1>>>(d_u, d_v, d_w, index, fx, fy, fz);   
  }
}


// Function to sum the total density
float sum_density() {
  float total_density = 0.0f;
  int size = (M + 2) * (N + 2) * (O + 2);
  for (int i = 0; i < size; i++) {
    total_density += dens[i];
  }
  return total_density;
}

// Simulation loop
void simulate(EventManager &eventManager, int timesteps) {
  //Copy data to GPU
  hipMemcpy(d_u, u, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v, v, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_w, w, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_u_prev, u_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v_prev, v_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_w_prev, w_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dens, dens, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dens_prev, dens_prev, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyHostToDevice);

  for (int t = 0; t < timesteps; t++) {
    // Get the events for the current timestep
    std::vector<Event> events = eventManager.get_events_at_timestamp(t);

    // Apply events to the simulation
    apply_events(events);

    // Perform the simulation steps
    vel_step(M, N, O, d_u, d_v, d_w, d_u_prev, d_v_prev, d_w_prev, visc, dt);
    dens_step(M, N, O, d_dens, d_dens_prev, d_u, d_v, d_w, diff, dt);
  }

  // Copy data back to CPU
  hipMemcpy(dens, d_dens, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyDeviceToHost);
}

int main() {
  // Initialize EventManager
  EventManager eventManager;
  eventManager.read_events("events.txt");

  // Get the total number of timesteps from the event file
  int timesteps = eventManager.get_total_timesteps();

  // Allocate and clear data
  if (!allocate_data())
    return -1;
  clear_data();

  // Run simulation with events
  simulate(eventManager, timesteps);

  // Print total density at the end of simulation
  float total_density = sum_density();
  std::cout << "Total density after " << timesteps
            << " timesteps: " << total_density << std::endl;

  // Free memory
  free_data();

  return 0;
}