#include "hip/hip_runtime.h"
#include "EventManager.h"
#include "fluid_solver.h"

#include <iostream>
#include <vector>

#define SIZE 84

#define IX(i, j, k) ((i) + (M + 2) * ((j) + (N + 2) * (k)))

#define ALIGNED_ARRAY_FLOAT(s,a) static_cast<float*>(std::aligned_alloc(a, s * sizeof(float)))

// Globals for the grid size
static int M = SIZE;
static int N = SIZE;
static int O = SIZE;
static float dt = 0.1f;      // Time delta
static float diff = 0.0001f; // Diffusion constant
static float visc = 0.0001f; // Viscosity constant

// Fluid simulation arrays
static float *u, *v, *w, *u_prev, *v_prev, *w_prev;
static float *dens, *dens_prev;

// GPU Arrays
static float *d_u, *d_v, *d_w, *d_u_prev, *d_v_prev, *d_w_prev;
static float *d_dens, *d_dens_prev;

// Function to allocate simulation data
int allocate_data() {
  int size = (M + 2) * (N + 2) * (O + 2);
  u = new float[size];
  v = new float[size];
  w = new float[size];
  u_prev = new float[size];
  v_prev = new float[size];
  w_prev = new float[size];
  dens = new float[size];
  dens_prev = new float[size];
  if (!u || !v || !w || !u_prev || !v_prev || !w_prev || !dens || !dens_prev) {
    std::cerr << "Cannot allocate memory" << std::endl;
    return 0;
  }

  // Allocate GPU memory
  hipMalloc(&d_u, size * sizeof(float));
  hipMalloc(&d_v, size * sizeof(float));
  hipMalloc(&d_w, size * sizeof(float));
  hipMalloc(&d_u_prev, size * sizeof(float));
  hipMalloc(&d_v_prev, size * sizeof(float));
  hipMalloc(&d_w_prev, size * sizeof(float));
  hipMalloc(&d_dens, size * sizeof(float));
  hipMalloc(&d_dens_prev, size * sizeof(float));

  return 1;
}
// Function to clear the data (set all to zero)
void clear_data() {
  int size = (M + 2) * (N + 2) * (O + 2);
  for (int i = 0; i < size; i++) {
    u[i] = v[i] = w[i] = u_prev[i] = v_prev[i] = w_prev[i] = dens[i] =
        dens_prev[i] = 0.0f;
  }

  // Copy data to GPU
  hipMemcpy(d_u, u, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v, v, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_w, w, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_u_prev, u_prev, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v_prev, v_prev, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_w_prev, w_prev, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dens, dens, size * sizeof(float), hipMemcpyHostToDevice); 
  hipMemcpy(d_dens_prev, dens_prev, size * sizeof(float), hipMemcpyHostToDevice);

}
// Free allocated memory
void free_data() {
  delete[] u;
  delete[] v;
  delete[] w;
  delete[] u_prev;
  delete[] v_prev;
  delete[] w_prev;
  delete[] dens;
  delete[] dens_prev;

  hipFree(d_u);
  hipFree(d_v);
  hipFree(d_w);
  hipFree(d_u_prev);
  hipFree(d_v_prev);
  hipFree(d_w_prev);
  hipFree(d_dens);
  hipFree(d_dens_prev);

}

__global__
void set_dens_kernel(float *d, float density, int index) {
  d[index] = density;
}

__global__
void set_forces_kernel(float *x, float *y, float *z, float fx, float fy, float fz, int index) {
  x[index] = fx;
  y[index] = fy;
  z[index] = fz;
}

// Apply events (source or force) for the current timestep
void apply_events(const std::vector<Event>& events) {
    int i = M / 2, j = N / 2, k = O / 2;
    int index = IX(i, j, k);
    bool density_set = false, force_set = false;
    float density = 0.0f, fx = 0.0f, fy = 0.0f, fz = 0.0f;

    for (const auto& event : events) {
        if (event.type == ADD_SOURCE) {
            density = event.density;
            density_set = true;
        } else if (event.type == APPLY_FORCE) {
            fx = event.force.x;
            fy = event.force.y;
            fz = event.force.z;
            force_set = true;
        }
    }

    if (density_set) {
        set_dens_kernel<<<1, 1>>>(d_dens, density, index);
    }

    if (force_set) {
        set_forces_kernel<<<1, 1>>>(d_u, d_v, d_w, fx, fy, fz, index);
    }
}


// Function to sum the total density
float sum_density() {
  float total_density = 0.0f;
  int size = (M + 2) * (N + 2) * (O + 2);
  for (int i = 0; i < size; i++) {
    total_density += dens[i];
  }
  return total_density;
}

// Simulation loop
void simulate(EventManager &eventManager, int timesteps) {
  for (int t = 0; t < timesteps; t++) {
    // Get the events for the current timestep
    std::vector<Event> events = eventManager.get_events_at_timestamp(t);

    // Apply events to the simulation
    apply_events(events);

    // Perform the simulation steps
    vel_step(M, N, O, d_u, d_v, d_w, d_u_prev, d_v_prev, d_w_prev, visc, dt);
    dens_step(M, N, O, d_dens, d_dens_prev, d_u, d_v, d_w, diff, dt);
  }

  // Copy dens back to CPU
  hipMemcpy(dens, d_dens, (M + 2) * (N + 2) * (O + 2) * sizeof(float), hipMemcpyDeviceToHost);
}

int main() {
  // Initialize EventManager
  EventManager eventManager;
  eventManager.read_events("events.txt");

  // Get the total number of timesteps from the event file
  int timesteps = eventManager.get_total_timesteps();

  // Allocate and clear data
  if (!allocate_data())
    return -1;
  clear_data();

  // Run simulation with events
  simulate(eventManager, timesteps);

  // Print total density at the end of simulation
  float total_density = sum_density();
  std::cout << "Total density after " << timesteps
            << " timesteps: " << total_density << std::endl;

  // Free memory
  free_data();

  return 0;
}